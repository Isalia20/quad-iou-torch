#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright [2024] Irakli Salia
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ******************************************************************************/

#define QUAD_ELEMENTS 8
#define MAX_INTERSECTION_POINTS 8
#define MAX_INSIDE_POINTS 8
#define MAX_ALL_POINTS 16
#define THREAD_COUNT_X 16
#define THREAD_COUNT_Y 16
#include <torch/extension.h>
#include <cmath>
#include "polygonArea.cuh"
#include "insidePoints.cuh"
#include "intersectionPoints.cuh"
#include "sortPoints.cuh"
#include "allPoints.cuh"
#include "simpleIntersectCheck.cuh"
#include "checks.cuh"


template <typename scalar_t>
__device__ inline scalar_t intersectionArea(
    const scalar_t *quad_0,
    const scalar_t *quad_1
) {
    // If we know that quad_0 and quad_1 are not
    // intersecting even a tiny bit(minimum enclosing box check)
    // we can skip below calculation altogether
    if (!simpleIntersectCheck::checkSimpleIntersection(quad_0, quad_1)) return 0.0;

    scalar_t intersection_points[MAX_INTERSECTION_POINTS][2];
    scalar_t inside_points[MAX_INSIDE_POINTS][2];
    scalar_t all_points[MAX_ALL_POINTS][2];
    allPoints::fillPointsWithInfinity(intersection_points,
                                      inside_points,
                                      all_points);

    intersectionPoints::findIntersectionPoints(quad_0,
                                               quad_1,
                                               intersection_points);
    insidePoints::findPointsInside(quad_0, quad_1, inside_points);
    allPoints::copyIntersectionInsidePoints(intersection_points,
                                            inside_points,
                                            all_points);
    sortPoints::sortPointsClockwise(all_points);
    scalar_t intersectArea = polygonArea::calcPolygonArea(all_points);
    return intersectArea;
}

template <typename scalar_t>
__device__ inline scalar_t unionArea(int quad_0_idx,
                                     int quad_1_idx,
                                     int quad_0_size,
                                     scalar_t *polygonAreas,
                                     scalar_t intersectArea) {
    return polygonAreas[quad_0_idx] + \
                polygonAreas[quad_0_size + quad_1_idx] - \
                    intersectArea;
}

template <typename scalar_t>
__device__ inline scalar_t calculateIoU(
    const scalar_t quad_0[QUAD_ELEMENTS],
    const scalar_t quad_1[QUAD_ELEMENTS],
    int quad_0_idx,
    int quad_1_idx,
    int quad_0_size,
    scalar_t *polygonAreas) {

    const scalar_t epsilon = 0.00001;

    scalar_t intersect_area = intersectionArea(quad_0, quad_1);
    return intersect_area / (unionArea(quad_0_idx, quad_1_idx, quad_0_size, polygonAreas, intersect_area) + epsilon);
}

template <typename scalar_t>
__global__ void calculateIoUKernel(
    scalar_t *quad_0,
    scalar_t *quad_1,
    scalar_t *iou_matrix,
    scalar_t *polygonAreas,
    int quad_0_size,
    int quad_1_size
    ) {
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    const int idx2 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Shared memory for storing quads
    __shared__ scalar_t quad_0_shared[THREAD_COUNT_X][QUAD_ELEMENTS];
    __shared__ scalar_t quad_1_shared[THREAD_COUNT_Y][QUAD_ELEMENTS];
    if (tx < QUAD_ELEMENTS && idx2 < quad_1_size) {
        quad_1_shared[ty][tx] = quad_1[idx2 * QUAD_ELEMENTS + tx];
    }
    if (ty < QUAD_ELEMENTS && idx1 < quad_0_size) {
        quad_0_shared[tx][ty] = quad_0[idx1 * QUAD_ELEMENTS + ty];
    }
    __syncthreads();

    if ((idx1 < quad_0_size) && (idx2 < quad_1_size)) {
        iou_matrix[idx1 * quad_1_size + idx2] = calculateIoU(quad_0_shared[tx],
                                                             quad_1_shared[ty],
                                                             idx1,
                                                             idx2,
                                                             quad_0_size,
                                                             polygonAreas);
    }
}


template <typename scalar_t>
__global__ void polygonAreaCalculationKernel(
    scalar_t *polygonAreas,
    scalar_t *quad_0,
    scalar_t *quad_1,
    int quad_0_size,
    int quad_1_size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < quad_0_size) {
        // Sort the points first since we are using gaussian formula
        scalar_t *quadrilateral = &quad_0[idx * 4 * 2];
        sortPoints::sortQuadPointsClockwise(quadrilateral);
        polygonAreas[idx] = polygonArea::calcQuadrilateralArea(quadrilateral);
    } else if (idx < (quad_0_size + quad_1_size)) {
        scalar_t *quadrilateral = &quad_1[(idx - quad_0_size) * 4 * 2];
        sortPoints::sortQuadPointsClockwise(quadrilateral);
        polygonAreas[idx] = polygonArea::calcQuadrilateralArea(quadrilateral);
    }
}

torch::Tensor calculateIoUCudaTorch(torch::Tensor quad_0, torch::Tensor quad_1) {
    checks::check_tensor_validity(quad_0, quad_1);
    // Create an output tensor
    torch::Tensor iou_matrix = torch::empty({quad_0.size(0), quad_1.size(0)}, quad_0.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(quad_0.scalar_type(), "calculateIoUCudaTorch", ([&] {
        scalar_t *polygonAreas_d;
        hipMalloc((void**)&polygonAreas_d, (quad_0.size(0) + quad_1.size(0)) * sizeof(scalar_t));

        dim3 blockSizeQuad(128, 1, 1);
        dim3 gridSizeQuad((quad_0.size(0) + quad_1.size(0) + blockSizeQuad.x - 1) / blockSizeQuad.x, 1, 1);

        polygonAreaCalculationKernel<scalar_t><<<gridSizeQuad, blockSizeQuad>>>(
            polygonAreas_d,
            quad_0.data_ptr<scalar_t>(),
            quad_1.data_ptr<scalar_t>(),
            quad_0.size(0),
            quad_1.size(0));
        hipDeviceSynchronize();

        dim3 blockSize(THREAD_COUNT_X, THREAD_COUNT_Y);
        dim3 gridSize((quad_0.size(0) + blockSize.x - 1) / blockSize.x,
                        (quad_1.size(0) + blockSize.y - 1) / blockSize.y);
        calculateIoUKernel<scalar_t><<<gridSize, blockSize>>>(
            quad_0.data_ptr<scalar_t>(),
            quad_1.data_ptr<scalar_t>(),
            iou_matrix.data_ptr<scalar_t>(),
            polygonAreas_d,
            quad_0.size(0),
            quad_1.size(0));
        hipDeviceSynchronize();
        hipFree(polygonAreas_d);
    }));
    return iou_matrix;
}
